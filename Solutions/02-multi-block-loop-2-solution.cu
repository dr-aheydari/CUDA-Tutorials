
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void loop()
{
    int curr_loc = threadIdx.x + blockIdx.x * blockDim.x;
    
    printf("This is iteration number %d\n", curr_loc);
}

int main()
{
  /*
   * we could also do <<<1,10>>> or <<<5,2>>>
   */

  int threads = 5;
  int blocks = 2;
  
  loop<<<blocks, threads>>>();
  hipDeviceSynchronize();
}
